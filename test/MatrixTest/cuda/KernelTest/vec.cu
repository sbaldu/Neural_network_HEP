#include "hip/hip_runtime.h"

#include <cmath>
#include <cstdint>
#include <numeric>
#include <vector>

#include "../../../../src/cuda/DataFormats/VectorKernels.h"

#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN
#include "doctest.h"

TEST_CASE("Test vec_add_1") {
  const int N{10};
  std::vector<int> v1(N), v2(N), sum(N);
  std::iota(v1.begin(), v1.end(), 1.);
  std::iota(v2.begin(), v2.end(), 1.);

  const int size{sizeof(int) * N};
  int *d_v1, *d_v2, *d_sum;
  hipMalloc(&d_v1, size);
  hipMalloc(&d_v2, size);
  hipMalloc(&d_sum, size);

  hipMemcpy(d_v1, v1.data(), size, hipMemcpyHostToDevice);
  hipMemcpy(d_v2, v2.data(), size, hipMemcpyHostToDevice);
  vec_add<<<std::ceil(N/(float)(256)), 256>>>(d_v1, d_v2, d_sum, N);
  hipMemcpy(sum.data(), d_sum, size, hipMemcpyDeviceToHost);

  for (int i{}; i < N; ++i) {
	CHECK(sum[i] == 2*(i+1));
  }

  hipFree(d_v1);
  hipFree(d_v2);
  hipFree(d_sum);
}

TEST_CASE("Test vec_add_2") {
  const int N{10};
  std::vector<int> v1(N), v2(N);
  std::iota(v1.begin(), v1.end(), 1.);
  std::iota(v2.begin(), v2.end(), 1.);

  const int size{sizeof(int) * N};
  int *d_v1, *d_v2;
  hipMalloc(&d_v1, size);
  hipMalloc(&d_v2, size);

  hipMemcpy(d_v1, v1.data(), size, hipMemcpyHostToDevice);
  hipMemcpy(d_v2, v2.data(), size, hipMemcpyHostToDevice);
  vec_add<<<std::ceil(N/(float)(256)), 256>>>(d_v1, d_v2, N);
  hipMemcpy(v1.data(), d_v1, size, hipMemcpyDeviceToHost);

  for (int i{}; i < N; ++i) {
	CHECK(v1[i] == 2*(i+1));
  }

  hipFree(d_v1);
  hipFree(d_v2);
}

TEST_CASE("Test vec_sub_1") {
  const int N{10};
  std::vector<int> v1(N), v2(N), sum(N);
  std::iota(v1.begin(), v1.end(), N);
  std::iota(v2.begin(), v2.end(), 0.);

  const int size{sizeof(int) * N};
  int *d_v1, *d_v2, *d_sum;
  hipMalloc(&d_v1, size);
  hipMalloc(&d_v2, size);
  hipMalloc(&d_sum, size);

  hipMemcpy(d_v1, v1.data(), size, hipMemcpyHostToDevice);
  hipMemcpy(d_v2, v2.data(), size, hipMemcpyHostToDevice);
  vec_sub<<<std::ceil(N/(float)(256)), 256>>>(d_v1, d_v2, d_sum, N);
  hipMemcpy(sum.data(), d_sum, size, hipMemcpyDeviceToHost);

  for (int i{}; i < N; ++i) {
	CHECK(sum[i] == N);
  }

  hipFree(d_v1);
  hipFree(d_v2);
  hipFree(d_sum);
}

TEST_CASE("Test vec_sub_2") {
  const int N{10};
  std::vector<int> v1(N), v2(N);
  std::iota(v1.begin(), v1.end(), N);
  std::iota(v2.begin(), v2.end(), 0.);

  const int size{sizeof(int) * N};
  int *d_v1, *d_v2;
  hipMalloc(&d_v1, size);
  hipMalloc(&d_v2, size);

  hipMemcpy(d_v1, v1.data(), size, hipMemcpyHostToDevice);
  hipMemcpy(d_v2, v2.data(), size, hipMemcpyHostToDevice);
  vec_sub<<<std::ceil(N/(float)(256)), 256>>>(d_v1, d_v2, N);
  hipMemcpy(v1.data(), d_v1, size, hipMemcpyDeviceToHost);

  for (int i{}; i < N; ++i) {
	CHECK(v1[i] == N);
  }

  hipFree(d_v1);
  hipFree(d_v2);
}

TEST_CASE("Test vec_multiply") {
  const int N{10};
  std::vector<int> v(N);
  std::iota(v.begin(), v.end(), 0.);
  const int constant{5};

  const int size{sizeof(int) * N};
  int *d_v;
  hipMalloc(&d_v, size);

  hipMemcpy(d_v, v.data(), size, hipMemcpyHostToDevice);
  vec_multiply<<<std::ceil(N/(float)(256)), 256>>>(d_v, constant, N);
  hipMemcpy(v.data(), d_v, size, hipMemcpyDeviceToHost);

  for (int i{}; i < N; ++i) {
	CHECK(v[i] == constant * i);
  }

  hipFree(d_v);
}

TEST_CASE("Test vec_divide") {
  const int N{10};
  std::vector<double> v(N);
  std::iota(v.begin(), v.end(), 0.);
  const double constant{5.};

  const int size{sizeof(double) * N};
  double *d_v;
  hipMalloc(&d_v, size);

  hipMemcpy(d_v, v.data(), size, hipMemcpyHostToDevice);
  vec_divide<<<std::ceil(N/(float)(256)), 256>>>(d_v, constant, N);
  hipMemcpy(v.data(), d_v, size, hipMemcpyDeviceToHost);

  for (int i{}; i < N; ++i) {
	CHECK(v[i] == i / constant);
  }

  hipFree(d_v);
}
